#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdio.h>
#include <fstream>
#include <initializer_list>
#include <iostream>
#include <limits>
#include <vector>

#define CHECK_CUDNN(expression)                                                       \
  do {                                                                                \
    hipdnnStatus_t status = (expression);                                              \
    if (status != HIPDNN_STATUS_SUCCESS) {                                             \
      printf("[%s:%d] Error: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(status)); \
      exit(1);                                                                        \
    }                                                                                 \
  } while (0)

inline std::vector<float> generate_cpu_data(size_t size) {
  std::vector<float> ret(size, 1.0f);
  return ret;
}

inline void print_vector(const std::vector<float>& vec, size_t line_size, std::ostream& o_stream) {
  for (size_t i = 0; i < vec.size(); i++) {
    if (i % line_size == 0 && i != 0) {
      o_stream << "\n";
    }
    o_stream << vec[i] << " ";
  }
  return;
}

inline void print_dims(int64_t* dims) {
  std::cout << *dims << "," << *(dims + 1) << "," << *(dims + 2) << "," << *(dims + 3);
  return;
}

cudnnBackendDescriptor_t GetTensorDescriptor(const std::initializer_list<int64_t>& dims, int64_t id,
                                             bool is_virtual = false) {
  if (dims.size() != 4) {
    printf("[Error] [%s:%d] cudnn Backend API only support dims == 4\n", __FILE__, __LINE__);
    exit(1);
  }
  cudnnBackendDescriptor_t tensor_desc;
  hipdnnDataType_t dtype = HIPDNN_DATA_FLOAT;
  int64_t alignment = 4;
  int64_t* dims_ptr = nullptr;
#if defined(__cplusplus) && __cplusplus >= 201703L
  dims_ptr = dims.data();
#else
  std::vector<int64_t> dims_vec(dims.begin(), dims.end());
  dims_ptr = dims_vec.data();
#endif
  std::vector<int64_t> strides(dims.begin(), dims.end());
  // NCHW
  strides[0] = strides[1] * strides[2] * strides[3];
  strides[1] = strides[2] * strides[3];
  strides[2] = strides[3];
  strides[3] = 1;

  // NHWC
  // std::vector<int64_t> dims_vec_for_nhwc(dims.begin(), dims.end());
  // strides[0] = dims_vec_for_nhwc[1] * dims_vec_for_nhwc[2] * dims_vec_for_nhwc[3];
  // strides[1] = 1;
  // strides[2] = dims_vec_for_nhwc[1] * dims_vec_for_nhwc[3];
  // strides[3] = dims_vec_for_nhwc[2];
  CHECK_CUDNN(cudnnBackendCreateDescriptor(CUDNN_BACKEND_TENSOR_DESCRIPTOR, &tensor_desc));
  CHECK_CUDNN(cudnnBackendSetAttribute(tensor_desc, CUDNN_ATTR_TENSOR_DATA_TYPE, CUDNN_TYPE_DATA_TYPE, 1, &dtype));
  CHECK_CUDNN(cudnnBackendSetAttribute(tensor_desc, CUDNN_ATTR_TENSOR_DIMENSIONS, CUDNN_TYPE_INT64, 4, dims_ptr));
  CHECK_CUDNN(cudnnBackendSetAttribute(tensor_desc, CUDNN_ATTR_TENSOR_STRIDES, CUDNN_TYPE_INT64, 4, strides.data()));
  CHECK_CUDNN(cudnnBackendSetAttribute(tensor_desc, CUDNN_ATTR_TENSOR_UNIQUE_ID, CUDNN_TYPE_INT64, 1, &id));
  CHECK_CUDNN(cudnnBackendSetAttribute(tensor_desc, CUDNN_ATTR_TENSOR_BYTE_ALIGNMENT, CUDNN_TYPE_INT64, 1, &alignment));
  if (is_virtual) {
    CHECK_CUDNN(
        cudnnBackendSetAttribute(tensor_desc, CUDNN_ATTR_TENSOR_IS_VIRTUAL, CUDNN_TYPE_BOOLEAN, 1, &is_virtual));
  }
  CHECK_CUDNN(cudnnBackendFinalize(tensor_desc));
  std::cout << (char)id << " dims:";
  print_dims(dims_ptr);
  std::cout << " strides:";
  print_dims(strides.data());
  if (is_virtual) {
    std::cout << " Virtual";
  }
  std::cout << std::endl;

  return tensor_desc;
}

cudnnBackendDescriptor_t GetConvOp(cudnnBackendDescriptor_t input_desc, cudnnBackendDescriptor_t w_desc,
                                   cudnnBackendDescriptor_t output_desc, const std::initializer_list<int64_t>& pads,
                                   const std::initializer_list<int64_t>& strides) {
  if (pads.size() != 2 || strides.size() != 2) {
    printf("[Error] [%s:%d] cudnn Backend API only support pads.dim == 2 && strides.dim == 2\n", __FILE__, __LINE__);
    exit(1);
  }

  // set convolution descriptor
  cudnnBackendDescriptor_t conv_desc;
  int64_t conv_dim = 2;
  hipdnnDataType_t conv_dtype = HIPDNN_DATA_FLOAT;
  hipdnnConvolutionMode_t conv_mode = HIPDNN_CROSS_CORRELATION;
  int64_t dilation[] = {1, 1};

  int64_t* pads_ptr = nullptr;
  int64_t* strides_ptr = nullptr;
#if defined(__cplusplus) && __cplusplus >= 201703L
  pads_ptr = pads.data();
  strides_ptr = strides.data();
#else
  std::vector<int64_t> pads_vec(pads.begin(), pads.end());
  pads_ptr = pads_vec.data();
  std::vector<int64_t> strides_vec(strides.begin(), strides.end());
  strides_ptr = strides_vec.data();
#endif

  CHECK_CUDNN(cudnnBackendCreateDescriptor(CUDNN_BACKEND_CONVOLUTION_DESCRIPTOR, &conv_desc));
  CHECK_CUDNN(cudnnBackendSetAttribute(conv_desc, CUDNN_ATTR_CONVOLUTION_SPATIAL_DIMS, CUDNN_TYPE_INT64, 1, &conv_dim));
  CHECK_CUDNN(
      cudnnBackendSetAttribute(conv_desc, CUDNN_ATTR_CONVOLUTION_COMP_TYPE, CUDNN_TYPE_DATA_TYPE, 1, &conv_dtype));
  CHECK_CUDNN(cudnnBackendSetAttribute(conv_desc, CUDNN_ATTR_CONVOLUTION_CONV_MODE, CUDNN_TYPE_CONVOLUTION_MODE, 1,
                                       &conv_mode));
  CHECK_CUDNN(
      cudnnBackendSetAttribute(conv_desc, CUDNN_ATTR_CONVOLUTION_PRE_PADDINGS, CUDNN_TYPE_INT64, conv_dim, pads_ptr));
  CHECK_CUDNN(
      cudnnBackendSetAttribute(conv_desc, CUDNN_ATTR_CONVOLUTION_POST_PADDINGS, CUDNN_TYPE_INT64, conv_dim, pads_ptr));
  CHECK_CUDNN(
      cudnnBackendSetAttribute(conv_desc, CUDNN_ATTR_CONVOLUTION_DILATIONS, CUDNN_TYPE_INT64, conv_dim, dilation));
  CHECK_CUDNN(cudnnBackendSetAttribute(conv_desc, CUDNN_ATTR_CONVOLUTION_FILTER_STRIDES, CUDNN_TYPE_INT64, conv_dim,
                                       strides_ptr));
  CHECK_CUDNN(cudnnBackendFinalize(conv_desc));

  // set convolution fprop operation
  cudnnBackendDescriptor_t conv_op;
  float alpha = 1.0f;
  float beta = 0.0f;
  CHECK_CUDNN(cudnnBackendCreateDescriptor(CUDNN_BACKEND_OPERATION_CONVOLUTION_FORWARD_DESCRIPTOR, &conv_op));
  CHECK_CUDNN(cudnnBackendSetAttribute(conv_op, CUDNN_ATTR_OPERATION_CONVOLUTION_FORWARD_X,
                                       CUDNN_TYPE_BACKEND_DESCRIPTOR, 1, &input_desc));
  CHECK_CUDNN(cudnnBackendSetAttribute(conv_op, CUDNN_ATTR_OPERATION_CONVOLUTION_FORWARD_W,
                                       CUDNN_TYPE_BACKEND_DESCRIPTOR, 1, &w_desc));
  CHECK_CUDNN(cudnnBackendSetAttribute(conv_op, CUDNN_ATTR_OPERATION_CONVOLUTION_FORWARD_Y,
                                       CUDNN_TYPE_BACKEND_DESCRIPTOR, 1, &output_desc));
  CHECK_CUDNN(cudnnBackendSetAttribute(conv_op, CUDNN_ATTR_OPERATION_CONVOLUTION_FORWARD_CONV_DESC,
                                       CUDNN_TYPE_BACKEND_DESCRIPTOR, 1, &conv_desc));
  CHECK_CUDNN(
      cudnnBackendSetAttribute(conv_op, CUDNN_ATTR_OPERATION_CONVOLUTION_FORWARD_ALPHA, CUDNN_TYPE_FLOAT, 1, &alpha));
  CHECK_CUDNN(
      cudnnBackendSetAttribute(conv_op, CUDNN_ATTR_OPERATION_CONVOLUTION_FORWARD_BETA, CUDNN_TYPE_FLOAT, 1, &beta));
  CHECK_CUDNN(cudnnBackendFinalize(conv_op));
  return conv_op;
}

cudnnBackendDescriptor_t GetReluOp(cudnnBackendDescriptor_t input_desc, cudnnBackendDescriptor_t output_desc) {
  // set relu descriptor
  cudnnBackendDescriptor_t relu_desc;
  CHECK_CUDNN(cudnnBackendCreateDescriptor(CUDNN_BACKEND_POINTWISE_DESCRIPTOR, &relu_desc));
  cudnnPointwiseMode_t relu_mode = CUDNN_POINTWISE_RELU_FWD;
  hipdnnDataType_t dtype = HIPDNN_DATA_FLOAT;
  CHECK_CUDNN(cudnnBackendSetAttribute(relu_desc, CUDNN_ATTR_POINTWISE_MODE, CUDNN_TYPE_POINTWISE_MODE, 1, &relu_mode));
  CHECK_CUDNN(cudnnBackendSetAttribute(relu_desc, CUDNN_ATTR_POINTWISE_MATH_PREC, CUDNN_TYPE_DATA_TYPE, 1, &dtype));
  CHECK_CUDNN(cudnnBackendFinalize(relu_desc));

  // set relu operation
  cudnnBackendDescriptor_t relu_op;
  CHECK_CUDNN(cudnnBackendCreateDescriptor(CUDNN_BACKEND_OPERATION_POINTWISE_DESCRIPTOR, &relu_op));
  CHECK_CUDNN(cudnnBackendSetAttribute(relu_op, CUDNN_ATTR_OPERATION_POINTWISE_PW_DESCRIPTOR,
                                       CUDNN_TYPE_BACKEND_DESCRIPTOR, 1, &relu_desc));
  CHECK_CUDNN(cudnnBackendSetAttribute(relu_op, CUDNN_ATTR_OPERATION_POINTWISE_XDESC, CUDNN_TYPE_BACKEND_DESCRIPTOR, 1,
                                       &input_desc));
  CHECK_CUDNN(cudnnBackendSetAttribute(relu_op, CUDNN_ATTR_OPERATION_POINTWISE_YDESC, CUDNN_TYPE_BACKEND_DESCRIPTOR, 1,
                                       &output_desc));
  CHECK_CUDNN(cudnnBackendFinalize(relu_op));
  return relu_op;
}

cudnnBackendDescriptor_t GetPointwiseOp(const cudnnBackendDescriptor_t& first_input_desc,
                                        const cudnnBackendDescriptor_t& second_input_desc,
                                        const cudnnBackendDescriptor_t& output_desc, cudnnPointwiseMode_t mode) {
  // set pointwise descriptor
  if (mode != CUDNN_POINTWISE_ADD && mode != CUDNN_POINTWISE_MUL) {
    printf("[Error] cudnn Backend API pointwise only support mul & add\n");
    exit(1);
  }
  cudnnBackendDescriptor_t pointwise_desc;
  hipdnnDataType_t dtype = HIPDNN_DATA_FLOAT;
  CHECK_CUDNN(cudnnBackendCreateDescriptor(CUDNN_BACKEND_POINTWISE_DESCRIPTOR, &pointwise_desc));
  CHECK_CUDNN(cudnnBackendSetAttribute(pointwise_desc, CUDNN_ATTR_POINTWISE_MODE, CUDNN_TYPE_POINTWISE_MODE, 1, &mode));
  CHECK_CUDNN(
      cudnnBackendSetAttribute(pointwise_desc, CUDNN_ATTR_POINTWISE_MATH_PREC, CUDNN_TYPE_DATA_TYPE, 1, &dtype));
  CHECK_CUDNN(cudnnBackendFinalize(pointwise_desc));
  // set pointwise op
  cudnnBackendDescriptor_t pointwise_op;
  CHECK_CUDNN(cudnnBackendCreateDescriptor(CUDNN_BACKEND_OPERATION_POINTWISE_DESCRIPTOR, &pointwise_op));
  CHECK_CUDNN(cudnnBackendSetAttribute(pointwise_op, CUDNN_ATTR_OPERATION_POINTWISE_PW_DESCRIPTOR,
                                       CUDNN_TYPE_BACKEND_DESCRIPTOR, 1, &pointwise_desc));
  CHECK_CUDNN(cudnnBackendSetAttribute(pointwise_op, CUDNN_ATTR_OPERATION_POINTWISE_XDESC,
                                       CUDNN_TYPE_BACKEND_DESCRIPTOR, 1, &first_input_desc));
  CHECK_CUDNN(cudnnBackendSetAttribute(pointwise_op, CUDNN_ATTR_OPERATION_POINTWISE_BDESC,
                                       CUDNN_TYPE_BACKEND_DESCRIPTOR, 1, &second_input_desc));
  CHECK_CUDNN(cudnnBackendSetAttribute(pointwise_op, CUDNN_ATTR_OPERATION_POINTWISE_YDESC,
                                       CUDNN_TYPE_BACKEND_DESCRIPTOR, 1, &output_desc));
  CHECK_CUDNN(cudnnBackendFinalize(pointwise_op));
  return pointwise_op;
}

cudnnBackendDescriptor_t GetGraph(hipdnnHandle_t handle, cudnnBackendDescriptor_t* ops, int64_t len) {
  cudnnBackendDescriptor_t op_graph;
  CHECK_CUDNN(cudnnBackendCreateDescriptor(CUDNN_BACKEND_OPERATIONGRAPH_DESCRIPTOR, &op_graph));
  CHECK_CUDNN(
      cudnnBackendSetAttribute(op_graph, CUDNN_ATTR_OPERATIONGRAPH_OPS, CUDNN_TYPE_BACKEND_DESCRIPTOR, len, ops));
  CHECK_CUDNN(cudnnBackendSetAttribute(op_graph, CUDNN_ATTR_OPERATIONGRAPH_HANDLE, CUDNN_TYPE_HANDLE, 1, &handle));
  CHECK_CUDNN(cudnnBackendFinalize(op_graph));

  // debug info
  int64_t graph_support_engine_count = -1;
  CHECK_CUDNN(cudnnBackendGetAttribute(op_graph, CUDNN_ATTR_OPERATIONGRAPH_ENGINE_GLOBAL_COUNT, CUDNN_TYPE_INT64, 1,
                                       nullptr, &graph_support_engine_count));
  std::cout << "cudnn graph support engine count: " << graph_support_engine_count << std::endl;

  return op_graph;
}

cudnnBackendDescriptor_t GetEngineSearcher(cudnnBackendDescriptor_t op_graph) {
  cudnnBackendDescriptor_t heuristic_searcher;
  CHECK_CUDNN(cudnnBackendCreateDescriptor(CUDNN_BACKEND_ENGINEHEUR_DESCRIPTOR, &heuristic_searcher));
  CHECK_CUDNN(cudnnBackendSetAttribute(heuristic_searcher, CUDNN_ATTR_ENGINEHEUR_OPERATION_GRAPH,
                                       CUDNN_TYPE_BACKEND_DESCRIPTOR, 1, &op_graph));
  cudnnBackendHeurMode_t search_mode = CUDNN_HEUR_MODE_FALLBACK;
  CHECK_CUDNN(
      cudnnBackendSetAttribute(heuristic_searcher, CUDNN_ATTR_ENGINEHEUR_MODE, CUDNN_TYPE_HEUR_MODE, 1, &search_mode));
  CHECK_CUDNN(cudnnBackendFinalize(heuristic_searcher));
  return heuristic_searcher;
}

// input : (n, c, h, w)
// w     : (k, c, r, s)
// output: (n, k, h, w)
// pad = 1, r = s = 3, so that output(h, w) == input(h, w)
// c is input channel, k is output channel

int main() {
  std::cout << "CUDNN VERSION: " << CUDNN_MAJOR << "." << CUDNN_MINOR << std::endl;
  hipdnnHandle_t handle;
  CHECK_CUDNN(hipdnnCreate(&handle));

  int64_t n = 1, c = 32, h = 4, w = 4;
  int64_t k = 32, r = 1, s = 1;

  // set input descriptor
  cudnnBackendDescriptor_t input_desc = GetTensorDescriptor({n, c, h, w}, 'x');
  int64_t input_size = n * c * h * w;

  // set filter descriptor
  cudnnBackendDescriptor_t conv_weight_desc = GetTensorDescriptor({k, c, r, s}, 'w');
  int64_t conv_weight_size = k * c * r * s;

  // set output descriptor
  cudnnBackendDescriptor_t conv_output_desc = GetTensorDescriptor({n, k, h, w}, 'C', true);
  int64_t conv_output_size = n * k * h * w;

  cudnnBackendDescriptor_t z_desc = GetTensorDescriptor({n, k, h, w}, 'z');
  int64_t z_size = n * k * h * w;

  cudnnBackendDescriptor_t add_op1_output_desc = GetTensorDescriptor({n, k, h, w}, 'A', true);

  cudnnBackendDescriptor_t bias_desc = GetTensorDescriptor({1, k, 1, 1}, 'b');
  int64_t b_size = 1 * k * 1 * 1;

  cudnnBackendDescriptor_t add_op2_output_desc = GetTensorDescriptor({n, k, h, w}, 'B', true);

  cudnnBackendDescriptor_t output_desc = GetTensorDescriptor({n, k, h, w}, 'y');

  // set convolution operator
  cudnnBackendDescriptor_t fprop = GetConvOp(input_desc, conv_weight_desc, conv_output_desc, {0, 0}, {1, 1});

  cudnnBackendDescriptor_t add_op1 = GetPointwiseOp(conv_output_desc, z_desc, add_op1_output_desc, CUDNN_POINTWISE_ADD);

  cudnnBackendDescriptor_t add_op2 =
      GetPointwiseOp(add_op1_output_desc, bias_desc, add_op2_output_desc, CUDNN_POINTWISE_ADD);

  // set relu operator
  cudnnBackendDescriptor_t relu = GetReluOp(add_op2_output_desc, output_desc);

  // set ConvTranspose, use ConvolutionBackword for impl, dy is x, w is w, dx is y
  // cudnnBackendDescriptor_t dgrad;
  // CHECK_CUDNN(cudnnBackendCreateDescriptor(CUDNN_BACKEND_OPERATION_CONVOLUTION_BACKWARD_DATA_DESCRIPTOR,
  // &dgrad)); CHECK_CUDNN(cudnnBackendSetAttribute(dgrad, CUDNN_ATTR_OPERATION_CONVOLUTION_BWD_DATA_DY,
  //                                      CUDNN_TYPE_BACKEND_DESCRIPTOR, 1, &xDesc));
  // CHECK_CUDNN(cudnnBackendSetAttribute(dgrad, CUDNN_ATTR_OPERATION_CONVOLUTION_BWD_DATA_W,
  //                                      CUDNN_TYPE_BACKEND_DESCRIPTOR, 1, &wDesc));
  // CHECK_CUDNN(cudnnBackendSetAttribute(dgrad, CUDNN_ATTR_OPERATION_CONVOLUTION_BWD_DATA_DX,
  //                                      CUDNN_TYPE_BACKEND_DESCRIPTOR, 1, &afterAddDesc));
  // CHECK_CUDNN(cudnnBackendSetAttribute(dgrad, CUDNN_ATTR_OPERATION_CONVOLUTION_BWD_DATA_CONV_DESC,
  //                                      CUDNN_TYPE_BACKEND_DESCRIPTOR, 1, &cDesc));
  // CHECK_CUDNN(
  //     cudnnBackendSetAttribute(dgrad, CUDNN_ATTR_OPERATION_CONVOLUTION_BWD_DATA_ALPHA, CUDNN_TYPE_FLOAT, 1,
  //     &alpha));
  // CHECK_CUDNN(
  //     cudnnBackendSetAttribute(dgrad, CUDNN_ATTR_OPERATION_CONVOLUTION_BWD_DATA_BETA, CUDNN_TYPE_FLOAT, 1,
  //     &beta));
  // CHECK_CUDNN(cudnnBackendFinalize(dgrad));

  // set graph descriptor
  int64_t len = 4;
  cudnnBackendDescriptor_t ops[] = {fprop, add_op1, add_op2, relu};
  cudnnBackendDescriptor_t op_graph = GetGraph(handle, ops, len);

  // for search config
  cudnnBackendDescriptor_t heuristic_searcher = GetEngineSearcher(op_graph);

  cudnnBackendDescriptor_t engcfg1;
  CHECK_CUDNN(cudnnBackendCreateDescriptor(CUDNN_BACKEND_ENGINECFG_DESCRIPTOR, &engcfg1));
  int64_t config_count = 1;
  int64_t return_config_count = -1;
  CHECK_CUDNN(cudnnBackendGetAttribute(heuristic_searcher, CUDNN_ATTR_ENGINEHEUR_RESULTS, CUDNN_TYPE_BACKEND_DESCRIPTOR,
                                       config_count, &return_config_count, &engcfg1));
  std::cout << "return_config_count: " << return_config_count << std::endl;

  // set plan descriptor
  cudnnBackendDescriptor_t plan;
  CHECK_CUDNN(cudnnBackendCreateDescriptor(CUDNN_BACKEND_EXECUTION_PLAN_DESCRIPTOR, &plan));
  CHECK_CUDNN(cudnnBackendSetAttribute(plan, CUDNN_ATTR_EXECUTION_PLAN_ENGINE_CONFIG, CUDNN_TYPE_BACKEND_DESCRIPTOR, 1,
                                       &engcfg1));
  CHECK_CUDNN(cudnnBackendSetAttribute(plan, CUDNN_ATTR_EXECUTION_PLAN_HANDLE, CUDNN_TYPE_HANDLE, 1, &handle));
  CHECK_CUDNN(cudnnBackendFinalize(plan));
  int64_t workspaceSize;
  int64_t return_count;
  CHECK_CUDNN(cudnnBackendGetAttribute(plan, CUDNN_ATTR_EXECUTION_PLAN_WORKSPACE_SIZE, CUDNN_TYPE_INT64, 1,
                                       &return_count, &workspaceSize));

  // allooc device memory
  void* xData = nullptr;
  void* wData = nullptr;
  void* yData = nullptr;
  void* zData = nullptr;
  void* bData = nullptr;
  void* workspace = nullptr;
  hipMalloc(&xData, sizeof(float) * input_size);
  hipMalloc(&wData, sizeof(float) * conv_weight_size);
  hipMalloc(&yData, sizeof(float) * conv_output_size);
  hipMalloc(&zData, sizeof(float) * z_size);
  hipMalloc(&bData, sizeof(float) * b_size);
  hipMalloc(&workspace, workspaceSize);
  auto x_vec = generate_cpu_data(input_size);
  auto w_vec = generate_cpu_data(conv_weight_size);
  auto z_vec = generate_cpu_data(z_size);
  auto b_vec = generate_cpu_data(b_size);
  hipMemcpy(xData, x_vec.data(), sizeof(float) * input_size, hipMemcpyHostToDevice);
  hipMemcpy(wData, w_vec.data(), sizeof(float) * conv_weight_size, hipMemcpyHostToDevice);
  hipMemcpy(zData, z_vec.data(), sizeof(float) * z_size, hipMemcpyHostToDevice);
  hipMemcpy(bData, b_vec.data(), sizeof(float) * b_size, hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  // set data pack descriptor
  int data_count = 5;
  void* dev_ptrs[data_count] = {xData, wData, yData, zData, bData};  // device pointer
  int64_t uids[data_count] = {'x', 'w', 'y', 'z', 'b'};

  cudnnBackendDescriptor_t varpack;
  CHECK_CUDNN(cudnnBackendCreateDescriptor(CUDNN_BACKEND_VARIANT_PACK_DESCRIPTOR, &varpack));
  CHECK_CUDNN(cudnnBackendSetAttribute(varpack, CUDNN_ATTR_VARIANT_PACK_DATA_POINTERS, CUDNN_TYPE_VOID_PTR, data_count,
                                       dev_ptrs));
  CHECK_CUDNN(
      cudnnBackendSetAttribute(varpack, CUDNN_ATTR_VARIANT_PACK_UNIQUE_IDS, CUDNN_TYPE_INT64, data_count, uids));
  CHECK_CUDNN(cudnnBackendSetAttribute(varpack, CUDNN_ATTR_VARIANT_PACK_WORKSPACE, CUDNN_TYPE_VOID_PTR, 1, &workspace));
  CHECK_CUDNN(cudnnBackendFinalize(varpack));

  // exec
  CHECK_CUDNN(cudnnBackendExecute(handle, plan, varpack));

  hipDeviceSynchronize();

  std::vector<float> y_vec(conv_output_size, -1.0f);
  hipMemcpy(y_vec.data(), yData, sizeof(float) * conv_output_size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  std::fstream x_fs("x.txt", std::ios::out | std::ios::trunc);
  std::fstream y_fs("y.txt", std::ios::out | std::ios::trunc);
  std::fstream w_fs("w.txt", std::ios::out | std::ios::trunc);
  print_vector(y_vec, w, y_fs);
  print_vector(w_vec, s, w_fs);
  print_vector(x_vec, w, x_fs);

  hipFree(xData);
  hipFree(yData);
  hipFree(wData);
  hipFree(workspace);

  CHECK_CUDNN(hipdnnDestroy(handle));
  return 0;
}